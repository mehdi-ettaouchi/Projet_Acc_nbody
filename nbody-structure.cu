
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h> // drand48
#include <omp.h>

//#define DUMP

struct PointParticleType { 
  float x, y, z;
};

__global__ void MoveParticles(const int nParticles, struct PointParticleType* position, struct PointParticleType* speed, const float dt) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

  // Loop over particles that experience force
  float Fx = 0, Fy = 0, Fz = 0; 
  for (; i < nParticles; i += gridDim.x * blockDim.x) { 

    // Components of the gravity force on particle i
    // Loop over positions that exert force
    for (int j = 0, Fx = Fy = Fz = 0.; j < nParticles; j++) { 
      // No self interaction
      if (i != j) {
          // Avoid singularity and interaction with self
          const float softening = 1e-20;

          // Newton's law of universal gravity
          const float dx = position[j].x - position[i].x;
          const float dy = position[j].y - position[i].y;
          const float dz = position[j].z - position[i].z;
          const float drSquared  = dx*dx + dy*dy + dz*dz + softening;
          const float drPower32  = pow(drSquared, 3.0/2.0);
            
          // Calculate the net force
          Fx += dx / drPower32;  
          Fy += dy / drPower32;  
          Fz += dz / drPower32;
      }

    }

    // Accelerate particles in response to the gravitational force
    speed[i].x += dt*Fx; 
    speed[i].y += dt*Fy; 
    speed[i].z += dt*Fz;
  }

  // Move particles according to their velocities
  // O(N) work, so using a serial loop
  //#pragma acc parallel loop
  i = threadIdx.x + blockIdx.x * blockDim.x;
  for (; i < nParticles; i += gridDim.x * blockDim.x) { 
    position[i].x  += speed[i].x*dt;
    position[i].y  += speed[i].y*dt;
    position[i].z  += speed[i].z*dt;
  }
}

void dump(int iter, int nParticles, struct PointParticleType* position)
{
    char filename[64];
    snprintf(filename, 64, "output_cuda_%d.txt", iter);

    FILE *f;
    f = fopen(filename, "w+");

    int i;
    for (i = 0; i < nParticles; i++)
    {
        fprintf(f, "%e %e %e\n",
        position[i].x, position[i].y, position[i].z);
    }

    fclose(f);
}

int main(const int argc, const char** argv)
{

  // Problem size and other parameters
  const int nParticles = (argc > 1 ? atoi(argv[1]) : 16384);
  // Duration of test
  const int nSteps = (argc > 2)?atoi(argv[2]):10;
  // Particle propagation time step
  const float dt = 0.0005f;

  struct PointParticleType* position = (struct PointParticleType*)malloc(nParticles*sizeof(struct PointParticleType));
  struct PointParticleType* speed    = (struct PointParticleType*)malloc(nParticles*sizeof(struct PointParticleType));

  // Initialize random number generator and particles
  srand48(0x2020);

  int i;
  for (i = 0; i < nParticles; i++)
  {
    position[i].x =  2.0*drand48() - 1.0;
    position[i].y =  2.0*drand48() - 1.0;
    position[i].z =  2.0*drand48() - 1.0;
    speed[i].x    = 2.0*drand48() - 1.0;
    speed[i].y    = 2.0*drand48() - 1.0;
    speed[i].z    = 2.0*drand48() - 1.0;
  }
  
  // Perform benchmark
  printf("\nPropagating %d particles using 1 thread...\n\n", 
	 nParticles
	 );
  double rate = 0, dRate = 0; // Benchmarking data
  const int skipSteps = 3; // Skip first iteration (warm-up)
  printf("\033[1m%5s %10s %10s %8s\033[0m\n", "Step", "Time, s", "Interact/s", "GFLOP/s"); fflush(stdout);
  for (int step = 1; step <= nSteps; step++) {

    struct PointParticleType *d_position;
    struct PointParticleType *d_speed;
    
    size_t size = nParticles*sizeof(struct PointParticleType);
    hipMalloc(&d_position, size); hipMalloc(&d_speed, size);
    hipMemcpy(d_position, position, size, hipMemcpyHostToDevice);
    hipMemcpy(d_speed   , speed   , size, hipMemcpyHostToDevice);

    int threadPerBlocs = 256;
    /* Ceil */
    int blocksPerGrid   = (nParticles + threadPerBlocs - 1) / threadPerBlocs;

    const double tStart = omp_get_wtime(); // Start timing
    MoveParticles<<< blocksPerGrid, threadPerBlocs >>>(nParticles, d_position, d_speed, dt);
    const double tEnd = omp_get_wtime(); // End timing

    hipMemcpy(position, d_position, size, hipMemcpyDeviceToHost);
    hipMemcpy(speed   , d_speed   , size, hipMemcpyDeviceToHost);
    hipFree(d_position); hipFree(d_speed);

    const float HztoInts   = ((float)nParticles)*((float)(nParticles-1)) ;
    const float HztoGFLOPs = 20.0*1e-9*((float)(nParticles))*((float)(nParticles-1));

    if (step > skipSteps) { // Collect statistics
      rate  += HztoGFLOPs/(tEnd - tStart); 
      dRate += HztoGFLOPs*HztoGFLOPs/((tEnd - tStart)*(tEnd-tStart)); 
    }

    printf("%5d %10.3e %10.3e %8.1f %s\n", 
	   step, (tEnd-tStart), HztoInts/(tEnd-tStart), HztoGFLOPs/(tEnd-tStart), (step<=skipSteps?"*":""));
    fflush(stdout);

#ifdef DUMP
    dump(step, nParticles, position);
#endif
  }
  rate/=(double)(nSteps-skipSteps); 
  dRate=sqrt(dRate/(double)(nSteps-skipSteps)-rate*rate);
  printf("-----------------------------------------------------\n");
  printf("\033[1m%s %4s \033[42m%10.1f +- %.1f GFLOP/s\033[0m\n",
	 "Average performance:", "", rate, dRate);
  printf("-----------------------------------------------------\n");
  printf("* - warm-up, not included in average\n\n");
  free(position);
  free(speed);
  return 0;
}


